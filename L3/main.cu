#include "hip/hip_runtime.h"
#include <iostream>
#include "json.hpp"
#include <vector>
#include <fstream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

constexpr int RESULT_LENGTH = 366;
constexpr int MAX_NAME_LENGTH = 20;
constexpr int HASHED_STRING_LENGHT = 30;

using json = nlohmann::json;
using namespace std;

struct Person
{
    int id;
    double age;
    char name[MAX_NAME_LENGTH];
};

vector<Person> load_json_file(const string &file_name)
{
    ifstream f(file_name);
    if (!f.is_open())
    {
        cerr << "Failed to open given '" << file_name << "' file." << endl;
        return std::vector<Person>();
    }
    json data = json::parse(f);
    vector<Person> data_vector;
    for (auto &element : data)
    {
        Person person;
        element.at("id").get_to(person.id);
        element.at("age").get_to(person.age);
        string name_str = element.at("name");
        strncpy(person.name, name_str.c_str(), MAX_NAME_LENGTH);
        person.name[MAX_NAME_LENGTH - 1] = '\0';
        data_vector.push_back(person);
    }
    f.close();

    return data_vector;
}

__device__ void hash_person(const Person *person, unsigned char *hash)
{
    // printf("person name: %s, age: %f, id: %d\n", person->name, person->age, person->id);
    for (int i = 0; i < HASHED_STRING_LENGHT - 1; i++)
    {
        if (i < 15)
        {
            // the first 15 characters will be name hash
            int name = (int)pow((int)person->name % 25 * i, 2);
            char randomChar = (char)(name % 25 + 'A');
            hash[i] = randomChar;
        }
        else if (i > 15 && i < 23)
        {
            // 16th-23rd characters will be id hash
            char randomChar = (char)((int)pow(person->id * i, 2) % 25 + 'A');
            hash[i] = randomChar;
        }
        else if (i > 23)
        {
            // last characters will be age hash
            char randomChar = (char)((int)pow(person->age * i, 3) % 25 + 'A');
            hash[i] = randomChar;
        }
    }

    hash[HASHED_STRING_LENGHT - 1] = '\0';
}

__device__ bool hash_passes_test(unsigned char *hash)
{
    int frequency[256] = {0}; // Initialize all frequencies to 0

    for (int i = 0; i < HASHED_STRING_LENGHT; i++)
    {
        frequency[hash[i]]++; // Increment the frequency of the current character

        if (frequency[hash[i]] >= 6) // If the current character appears four times or more
        {
            return true;
        }
    }
    return false;
}

__global__ void calculate_hashes(const Person *persons, unsigned char *result, const int *persons_count, int *last_filled_person_index)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x; // total number of threads

    // printf("Starting calculating. Index: %d, stride: %d\n", index, stride);

    if (index < RESULT_LENGTH)
    {
        // printf("Calculating hashes for index: %d\n", index);
        for (int i = index; i < *persons_count; i += stride)
        {
            unsigned char hashed_string[HASHED_STRING_LENGHT];
            memset(hashed_string, ' ', HASHED_STRING_LENGHT);
            hash_person(&persons[i], hashed_string);
            hashed_string[HASHED_STRING_LENGHT - 1] = '\n';

            bool passed = hash_passes_test(hashed_string);
            printf("Hashed string: %s, passed: %d\n", hashed_string, passed);
            if (passed)
            {
                const int current_index = atomicAdd(last_filled_person_index, 1);
                // printf("last filled index: %d\n", current_index);
                memcpy(result + HASHED_STRING_LENGHT * current_index, hashed_string, HASHED_STRING_LENGHT);
            }
            else
            {
                printf("Skipped.\n");
            }
        }
    }
}

int main()
{
    // read data
    const string data_file_name = "data.json";
    const vector<Person> data = load_json_file(data_file_name);
    cout << "Loaded " << data.size() << " persons from '" << data_file_name << "'." << std::endl;
    const bool data_exists = data.size() > 0;
    if (!data_exists)
    {
        cerr << "There is no data in '" + data_file_name + "'. Closing the program." << std::endl;
        return 1;
    }

    int device;
    hipError_t error = hipGetDevice(&device);
    if (error)
    {
        cerr << "Encountered an error while getting device number. Error: " << error << endl;
        return 1;
    }

    // unsigned char *result = new unsigned char[RESULT_LENGTH * HASHED_STRING_LENGHT];
    unsigned char *result;
    error = hipMalloc(&result, RESULT_LENGTH * HASHED_STRING_LENGHT);
    if (error)
    {
        cerr << "Couldn't allocate memory for result on GPU. Error: " << error << endl;
        return 1;
    }

    hipDeviceProp_t prop;
    error = hipGetDeviceProperties(&prop, device);
    if (error)
    {
        cerr << "Error while getting CUDA " << device << " device properties. Error: " << error << endl;
        hipFree(result);
        return 1;
    }

    // const int MAX_THREADS_IN_BLOCK = prop.maxThreadsPerBlock;
    // const int MAX_THREADS_IN_GPU = prop.maxThreadsPerMultiProcessor;
    // const int MAX_BLOCKS_IN_GPU = prop.maxBlocksPerMultiProcessor;
    //
    // cout << "Max threads in block: " << MAX_THREADS_IN_BLOCK << endl; // 1024
    // cout << "Max threads in gpu: " << MAX_THREADS_IN_GPU << endl;     // 1536
    // cout << "Max blocks in gpu: " << MAX_BLOCKS_IN_GPU << endl;       // 16

    const int THREADS_IN_BLOCK = 32;
    const int BLOCK_COUNT = 2;

    const int PERSONS_COUNT = data.size();
    const int PERSONS_SIZE = PERSONS_COUNT * sizeof(Person);

    Person *d_persons;
    error = hipMalloc(&d_persons, PERSONS_SIZE);
    if (error)
    {
        cerr << "Couldn't allocate memory for persons on GPU. Error: " << error << endl;
        hipFree(result);
        return 1;
    }

    error = hipMemcpy(d_persons, data.data(), PERSONS_SIZE, hipMemcpyHostToDevice);
    if (error)
    {
        cerr << "Couldn't copy persons to GPU. Error: " << error << endl;
        hipFree(result);
        hipFree(d_persons);
        return 1;
    }

    int *d_persons_count;
    error = hipMalloc(&d_persons_count, sizeof(int));
    if (error)
    {
        cerr << "Couldn't allocate memory for persons count on GPU. Error: " << error << endl;
        hipFree(result);
        hipFree(d_persons);
        return 1;
    }

    error = hipMemcpy(d_persons_count, &PERSONS_COUNT, sizeof(int), hipMemcpyHostToDevice);
    if (error)
    {
        cerr << "Couldn't copy persons count to GPU. Error: " << error << endl;
        hipFree(result);
        hipFree(d_persons);
        hipFree(d_persons_count);
        return 1;
    }

    int *last_filled_persons_index;
    hipMalloc(&last_filled_persons_index, sizeof(int));
    hipMemcpy(last_filled_persons_index, 0, sizeof(int), hipMemcpyHostToDevice);
    calculate_hashes<<<BLOCK_COUNT, THREADS_IN_BLOCK>>>(d_persons, result, d_persons_count, last_filled_persons_index);

    error = hipDeviceSynchronize();
    if (error)
    {
        cerr << "Couldn't synchronise CPU with GPU. Error: " << error << endl;
        hipFree(result);
        hipFree(d_persons);
        hipFree(d_persons_count);
        return 1;
    }

    unsigned char *result_on_cpu = new unsigned char[RESULT_LENGTH * HASHED_STRING_LENGHT];
    error = hipMemcpy(result_on_cpu, result, RESULT_LENGTH * HASHED_STRING_LENGHT, hipMemcpyDeviceToHost);
    if (error)
    {
        cerr << "Couldn't copy result from GPU to CPU. Error: " << error << endl;
        hipFree(result);
        hipFree(d_persons);
        hipFree(d_persons_count);
        delete[] result_on_cpu;
        return 1;
    }

    result_on_cpu[RESULT_LENGTH * HASHED_STRING_LENGHT - 1] = '\0';

    if (remove("results.txt") == 0)
    {
        printf("File deleted successfully\n");
    }
    else
    {
        printf("Unable to delete the file or file doesn't exist\n");
    }

    ofstream out("results.txt");
    out << result_on_cpu;
    out.close();

    printf("Results: \n");
    int results_length = strlen((char *)result_on_cpu);
    int person_i = 1;
    for (int i = 0; i < results_length; i++)
    {

        if (result_on_cpu[i] == '\n')
        {
            printf("\n%d: ", person_i);
            person_i++;
        }
        else if (result_on_cpu[i] == '\0')
            printf("NULL");
        else
            printf("%c", result_on_cpu[i]);
    }

    delete[] result_on_cpu;
    hipFree(result);
    hipFree(d_persons);
    hipFree(d_persons_count);
    return 0;
}
